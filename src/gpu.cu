/*
 *  gpu.cpp
 *
 *  Author: imen chakroun
 *
 */

#include <malloc.h>
#include <pthread.h>
#include <semaphore.h>
#include <algorithm>
#include <stack>
#include <vector>
#include <iterator>
#include <list>

#include <time.h>
#include <sys/time.h>
#include <iostream>

#include <cutil.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

#include <shrUtils.h>

#include "../headers/pbab.h"
#include "../headers/peer.h"
#include "../headers/explorer.h"
#include "../headers/tree.h"
#include "../headers/weights.h"
#include "../headers/problems.h"

#include "../headers/bound_flowshop_gpu.h"

#include "../headers/gpu.h"
#include "../headers/bound_abstract.h"
#include "../headers/instance_abstract.h"

#include "../headers/solutions.h"

#include "bound_flowshop_gpu.cu"

#define BEGIN_ORDER 0
#define END_ORDER 1

bool descendingCom(problem const* n1, problem const* n2){
	return (n1->couts_somme >= n2->couts_somme);
}
hipError_t cudaerror_t;

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();

    if( hipSuccess != err)
    {
    	fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

problemPool::problemPool(){
	this->children_d = NULL;
	this->children_bounds = NULL;
	this->children_size = 0;
}

problemPool::~problemPool(){
}

gpu::gpu(pbab* pbb,int nb_block, int block_size, int used_device, char johnson,char temps, char divergence)
{
    this->johnson = johnson;
    this->temps = temps;
    this->divergence = divergence;
    this->pbb = pbb;
    this->block_size = block_size;
    this->nb_block = nb_block;
    this->used_device = used_device;

}

void gpu::init_tabs()
{
    cout << "Using this device " << used_device <<" " << nb_block <<" " << block_size << endl<<flush;

	switch (used_device)
	{
		case 0: { hipSetDevice(0); break; }
		case 1: { hipSetDevice(1); break; }
		case 2: { hipSetDevice(2);	break; }
		case 3:	{ hipSetDevice(3);	break; }
		default: cout << "No Cuda Capable Device : Error in gpu::init()" <<endl <<flush ; exit(0);
	}

//	int *bounds = (int *) malloc(nb_block * block_size* sizeof(int)) ;

//	for (int i = 0 ; i < nb_block * block_size ; i++) bounds[i] = 0;

//	pool_to_evaluate = (problem_d *) malloc(nb_block * block_size * sizeof(problem_d)) ;

//	for (int i = 0 ; i < nb_block * block_size; i++) init_problem_device_g(pool_to_evaluate[i]);

	// Create pinned memory in host CPU for ansynchronous copy
	for (int i=0; i<MAX_SIZE_INPUTQUEUE_TO_GPU; i++){

		// Asynchronous
		hipHostMalloc((void**) &pool_to_evaluate_h[i], pool_size * sizeof(raw_bb_problem));
		hipHostMalloc((void**) &bounds_h[i], pool_size * sizeof(int));

		hipStreamCreate(&stream[i]);

		// Allocate Memory in GPU device
		hipMalloc( (void**) &pool_to_evaluate_d[i], pool_size * sizeof(raw_bb_problem));
		hipMalloc( (void**) &bounds_d[i], nb_block * pool_size * sizeof(int));
	}
	current_index_gpu = 0;
	starting_index_gpu = 0;

//	hipMemcpy(bounds_d, bounds, nb_block * block_size * sizeof(int), hipMemcpyHostToDevice);
}

void gpu::init()
{
 	(pbb->instance->data)->seekg (0); 
	(pbb->instance->data)->clear ();

   	*(pbb->instance->data)>>nbJob_g;
	*(pbb->instance->data)>>nbMachines_g;

	*(pbb->instance->data)>>time_seed_g;

	tempsJob_g = (int *)malloc( nbMachines_g * nbJob_g * sizeof(int));
	tempsJob_T = (int **)malloc( nbMachines_g * sizeof(int *));

	for(int i = 0; i < nbMachines_g; i++)
	   tempsJob_T[i] = (int *)malloc( nbJob_g * sizeof(int)) ;

	for(int i = 0; i < nbMachines_g; i++)
	  for(int j = 0; j < nbJob_g; j++)
	  {
		  *(pbb->instance->data)>>tempsJob_T[i][j];
		   tempsJob_g[i * nbJob_g + j] = tempsJob_T[i][j];
	  }


	init_tabs();
	init_bound();

	allocate_on_device();

	checkCUDAError("allocation échouée");

	copy_to_device();

	checkCUDAError("copie échouée");

	free_memories();
}

void gpu::problem_to_problem_d(problem& pb,problem_d& pb_d)
{
	for (int l = 0; l < pbb->size; l++) pb_d.permutation[l] = pb.permutation[l];

	pb_d.limite1 = pb.limite1;

	pb_d.limite2 = pb.limite2;

	pb_d.depth = pb.depth;

	pb_d.begin_end = pb.begin_end;

	pb_d.father = (unsigned long long int) pb.father;

}

bool gpu::decompose_on_cpu(problems* fathers)
{
//	if (VERBOSE){
//		cout << statistics::time_get_in_second() << ": decompose_on_cpu, parent size: " << fathers->size() << endl;
//		cout << statistics::time_get_in_second() << ": decompose_on_cpu, parent size1: " << fathers->size1 << endl;
//	}
	

	problemPool *pp = new problemPool();

	raw_bb_problem* tmp = this->pool_to_evaluate_h[current_index_gpu] ;
	int index = -1;

	for (int l = 0; l < fathers->size() ; l++)
	{
		if (this->pbb->tr->bbCalculator->simple(fathers->at(l)))
			pbb->decompose->leaves_generate3(fathers->at(l),tmp, &index);
		else
		{
			pbb->decompose->problems_generate3(tmp,fathers->at(l),BEGIN_ORDER, &index);

			pbb->decompose->problems_generate3(tmp,fathers->at(l),END_ORDER, &index);
		}

	}

	if (index >= 0){
		pp->children_size = index + 1;
		pp->children_d = tmp;

		// Asynchronous Copy and launch in GPU
		cudaerror_t = hipMemcpyAsync(pool_to_evaluate_d[current_index_gpu], pp->children_d,
						pp->children_size * sizeof(raw_bb_problem), hipMemcpyHostToDevice, stream[current_index_gpu]);

		checkCUDAError("copie 2 échouée");

		Evaluate_ON_GPU<<<nb_block, block_size, 0, stream[current_index_gpu]>>>(pool_to_evaluate_d[current_index_gpu],
														bounds_d[current_index_gpu],
														nbJob_g,
														nbMachines_g,
														nbborne_g,
														somme_g,
														nbElem_g,
														nbFois_d,
														machine_d,
														tabJohnson_d,
														tempsJob_d,
														tempsLag_d,
														minTempsArr_d,
														ordoSomme_d,
														minTempsDep_d,
														pp->children_size,
														time_seed_g);
		cudaerror_t = hipMemcpyAsync(bounds_h[current_index_gpu], bounds_d[current_index_gpu], pp->children_size * sizeof(int),
							hipMemcpyDeviceToHost, stream[current_index_gpu]);

		checkCUDAError("copie 3 échouée");

		pbb->gp->readyPoolToGPU.push_back(pp);
		
		current_index_gpu = (current_index_gpu + 1) % MAX_SIZE_INPUTQUEUE_TO_GPU;

//		if (VERBOSE)
//			cout << statistics::time_get_in_second() << ": decompose_on_cpu, readyPool: " << pbb->gp->readyPoolToGPU.size() << endl;
		
		return true;
	}else{
//		free(pp->children_d);
		delete pp;
		
//		if (VERBOSE)
//			cout << statistics::time_get_in_second() << ": decompose_on_cpu, readyPool1: " << pbb->gp->readyPoolToGPU.size() << endl;
		
		return false;
	}
}

void gpu::prune(problemPool* pp){

//	if (VERBOSE){
//		cout << statistics::time_get_in_second() << ": Before prune, tree size: " << this->pbb->tr->size() << endl;
//		cout << statistics::time_get_in_second() << ": Before prune, num_elements: " << pp->children_size << endl;
//	}

	vector <problem*> fathers;
	int index = 0;
	while (index < pp->children_size){

			// assigned cost to b&b problems
			pp->children_d[index].couts_somme = pp->children_bounds[index];
			pp->children_d[index].couts[0] = pp->children_bounds[index];
			pp->children_d[index].couts[1] = 0;

			problem* father_problem = new problem(this->pbb, true);

			if (pp->children_d[index].begin_end == BEGIN_ORDER)
			{
				father_problem->debut.push_back((pp->children_d + index));
			}
			else
			{

				father_problem->fin.push_back((pp->children_d + index));

			}

			int consecutive_subproblems = (MAX_NBJOBS + 1 - pp->children_d[index].depth) * 2;
			consecutive_subproblems--;

			// continue add all subproblems to this parent
			for (int jj = 1; jj <= consecutive_subproblems; jj++){
				index++;

				// assigned cost to b&b problems
				pp->children_d[index].couts_somme = pp->children_bounds[index];
				pp->children_d[index].couts[0] = pp->children_bounds[index];
				pp->children_d[index].couts[1] = 0;

				if (pp->children_d[index].begin_end == BEGIN_ORDER)
				{
					father_problem->debut.push_back((pp->children_d + index));
				}
				else
				{
					father_problem->fin.push_back((pp->children_d + index));
				}

			}
			index++;

			// add this father problem to a list
			fathers.push_back(father_problem);
	}

//	if (VERBOSE)
//		cout << statistics::time_get_in_second() << ": prune, father size0: " << fathers.size() << endl;

	for(vector<problem*>::iterator i = fathers.begin(); i != fathers.end() ; ++i) //je parcours les pères
	{
		if ( pbb->decompose->choise( (*i)->debut,(*i)->fin ) == BEGIN_ORDER )
		{

			(*i)->debut.ranks();

			for(vector<raw_bb_problem*>::iterator j = (*i)->debut.begin(); j < (*i)->debut.end() ; ++j)
			{
				if(this->pbb->tr->bbCalculator->leaf(*j))
				{
					if (pbb->wsltns->insert(*j)){
						pbb->node->sendOptimalSolutionToGPUs(pbb);
						pbb->node->sendOptimalSolutionToCPUs(pbb);
						
//						if (VERBOSE){
//							cout << statistics::time_get_in_second() << ": cout: " << pbb->tr->cout << endl;
//							cout << statistics::time_get_in_second() << ": Optimal found: " << *(pbb->wsltns) << endl;
//						}
						
					}


					pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
				}
				else
				{
					if ( pbb->tr->bound(*j)){
						raw_bb_problem* new_problem = (raw_bb_problem*) malloc(sizeof(raw_bb_problem));
						memcpy(new_problem, *j, sizeof(raw_bb_problem));
						pbb->tr->insert(new_problem);
					}
					else {
						pbb->tr->prunedNodes++;
						pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
					}
				}
			}
		}
		else
		{

			(*i)->fin.ranks();
			for(vector<raw_bb_problem*>::iterator j = (*i)->fin.begin(); j < (*i)->fin.end() ; ++j)
			{

				if(this->pbb->tr->bbCalculator->leaf(*j))
				{

					if (pbb->wsltns->insert(*j)){
						pbb->node->sendOptimalSolutionToGPUs(pbb);
						pbb->node->sendOptimalSolutionToCPUs(pbb);

//						if (VERBOSE){
//							cout << statistics::time_get_in_second() << ": cout: " << pbb->tr->cout << endl;
//							cout << statistics::time_get_in_second() << ": Optimal found: " << *(pbb->wsltns) << endl;
//						}
						
					}

					pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
				}
				else
				{
					if ( pbb->wsltns->inserable(*j)){
						raw_bb_problem* new_problem = (raw_bb_problem*) malloc(sizeof(raw_bb_problem));
						memcpy(new_problem, *j, sizeof(raw_bb_problem));
						pbb->tr->insert(new_problem);
					}
					else {
						pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
					}
				}
			}
		}
		(*i)->debut.clear();
		(*i)->fin.clear();
	}

	for(vector<problem*>::iterator i = fathers.begin(); i != fathers.end() ; ++i) //je parcours les pères
	{
		delete (*i);
	}
	fathers.erase(fathers.begin(), fathers.end());

//	if (VERBOSE){
//		cout << statistics::time_get_in_second() << ": prune, father size: " << fathers.size() << endl;
//		cout << statistics::time_get_in_second() << ": prune, tree size: " << pbb->tr->size() << endl;
//		cout << statistics::time_get_in_second() << ": prune, tree size in bytes: " << pbb->tr->size() * sizeof(raw_bb_problem) << endl << endl;
//	}
	
}


void gpu::prune(raw_bb_problem* pp, int num_problems){

	if (VERBOSE){
		cout << statistics::time_get_in_second() << ": Before prune, tree size: " << this->pbb->tr->size() << endl;
		cout << statistics::time_get_in_second() << ": Before prune, num_elements: " << num_problems << endl;
	}

	vector <problem*> fathers;
	int index = 0;

	while (index < num_problems){

			problem* father_problem = new problem(this->pbb, true);

			if (pp[index].begin_end == BEGIN_ORDER)
			{
				father_problem->debut.push_back((pp + index));
			}
			else
			{

				father_problem->fin.push_back((pp + index));

			}

			int consecutive_subproblems = (MAX_NBJOBS + 1 - pp[index].depth) * 2;
			consecutive_subproblems--;

			// continue add all subproblems to this parent
			for (int jj = 1; jj <= consecutive_subproblems; jj++){
				index++;

				if (pp[index].begin_end == BEGIN_ORDER)
				{
					father_problem->debut.push_back((pp + index));
				}
				else
				{
					father_problem->fin.push_back((pp + index));
				}

			}
			index++;

			// add this father problem to a list
			fathers.push_back(father_problem);
	}
	
	if (VERBOSE)
		cout << statistics::time_get_in_second() << ": prune, father size0: " << fathers.size() << endl;

	for(vector<problem*>::iterator i = fathers.begin(); i != fathers.end() ; ++i) //je parcours les pères
	{
		if ( pbb->decompose->choise( (*i)->debut,(*i)->fin ) == BEGIN_ORDER )
		{

			(*i)->debut.ranks();

			for(vector<raw_bb_problem*>::iterator j = (*i)->debut.begin(); j < (*i)->debut.end() ; ++j)
			{
				
				if(this->pbb->tr->bbCalculator->leaf(*j))
				{

					if (pbb->wsltns->insert(*j)){
						pbb->node->sendOptimalSolutionToGPUs(pbb);
						pbb->node->sendOptimalSolutionToCPUs(pbb);

						if (VERBOSE){
							cout << statistics::time_get_in_second() << ": cout: " << pbb->tr->cout << endl;
							cout << statistics::time_get_in_second() << ": Optimal found: " << *(pbb->wsltns) << endl;
						}
						
					}

					pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
				}
				else
				{
					if ( pbb->tr->bound(*j)){

						raw_bb_problem* new_problem = (raw_bb_problem*) malloc(sizeof(raw_bb_problem));
						memcpy(new_problem, *j, sizeof(raw_bb_problem));
						pbb->tr->insert(new_problem);
					}
					else {
						pbb->tr->prunedNodes++;
						pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
					}
				}
			}
		}
		else
		{

			(*i)->fin.ranks();
			for(vector<raw_bb_problem*>::iterator j = (*i)->fin.begin(); j < (*i)->fin.end() ; ++j)
			{

				if(this->pbb->tr->bbCalculator->leaf(*j))
				{

					if (pbb->wsltns->insert(*j)){
						pbb->node->sendOptimalSolutionToGPUs(pbb);
						pbb->node->sendOptimalSolutionToCPUs(pbb);

						if (VERBOSE){
							cout << statistics::time_get_in_second() << ": cout: " << pbb->tr->cout << endl;
							cout << statistics::time_get_in_second() << ": Optimal found: " << *(pbb->wsltns) << endl;
						}
						
					}

					pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
				}
				else
				{
					if ( pbb->wsltns->inserable(*j)){

						raw_bb_problem* new_problem = (raw_bb_problem*) malloc(sizeof(raw_bb_problem));
						memcpy(new_problem, *j, sizeof(raw_bb_problem));
						pbb->tr->insert(new_problem);
					}
					else {
						pbb->tr->explored_leaves += pbb->wghts->depths[(*j)->depth];
					}
				}
			}
		}
//		(*i)->debut.clear();
//		(*i)->fin.clear();
	}

	for(vector<problem*>::iterator i = fathers.begin(); i != fathers.end() ; ++i)
	{
		delete (*i);
	}
	fathers.erase(fathers.begin(), fathers.end());

	if (VERBOSE){
		cout << statistics::time_get_in_second() << ": prune, father size: " << fathers.size() << endl;
		cout << statistics::time_get_in_second() << ": prune, tree size: " << pbb->tr->size() << endl;
		cout << statistics::time_get_in_second() << ": prune, tree size1 in bytes: " << pbb->tr->size() * sizeof(raw_bb_problem) << endl << endl;
	}
	
}
void gpu::prune_shareOutputGPU(problemPool* pp){
	if (VERBOSE)
		cout << statistics::time_get_in_second() << ": prune_shareOutputGPU, children size: " << pp->children_size << endl;
	
	int index = 0;
	while (index < pp->children_size){

			// assigned cost to b&b problems
			pp->children_d[index].couts_somme = pp->children_bounds[index];
			pp->children_d[index].couts[0] = pp->children_bounds[index];
			pp->children_d[index].couts[1] = 0;

			index++;
	}

	this->pbb->tr->outputGPUForShare.push_back(pp);
	
	if (VERBOSE)
		cout << statistics::time_get_in_second() << ": prune_shareOutputGPU, outputGPUForShare: " << this->pbb->tr->outputGPUForShare.size() << endl;

}
int* gpu::calculate_bounds_in_gpu(problemPool* pp){
	
//	if (VERBOSE)
//		cout << statistics::time_get_in_second() << ": GPU, calculate_bounds_in_gpu: " << pp->children_size << endl;
//
//	int *bounds = (int *) malloc(pp->children_size * sizeof(int)) ;
//
//	cudaerror_t = hipMemcpy(pool_to_evaluate_d, pp->children_d, pp->children_size * sizeof(raw_bb_problem), hipMemcpyHostToDevice);
//	checkCUDAError("copie 2 échouée");
//
//
//	Evaluate_ON_GPU<<<nb_block, block_size>>>(pool_to_evaluate_d,
//												bounds_d,
//												nbJob_g,
//												nbMachines_g,
//												nbborne_g,
//												somme_g,
//												nbElem_g,
//												nbFois_d,
//												machine_d,
//												tabJohnson_d,
//												tempsJob_d,
//												tempsLag_d,
//												minTempsArr_d,
//												ordoSomme_d,
//												minTempsDep_d,
//												pp->children_size,
//												time_seed_g);
//
//	cudaerror_t = hipMemcpy(bounds, bounds_d, pp->children_size * sizeof(int), hipMemcpyDeviceToHost);
//	checkCUDAError("copie 3 échouée");
//
//	if (VERBOSE){
//		cout << statistics::time_get_in_second() << ": GPU, return bounds: " << bounds[0] << endl;
//		cout << statistics::time_get_in_second() << ": GPU, outputPoolOfGPU: " << this->pbb->gp->outputPoolOfGPU.size() << endl;
//		cout << statistics::time_get_in_second() << ": GPU, outputGPUForShare: " << this->pbb->tr->outputGPUForShare.size() << endl;
//		cout << statistics::time_get_in_second() << ": GPU, Worker state: " << this->pbb->workerState << endl;
//	}
//
//	return bounds;
}

void gpu::allocate_on_device()
{
    hipMalloc( (void**) &tempsJob_d, nbJob_g * nbMachines_g * sizeof(int));
    hipMalloc( (void**) &tabJohnson_d, nbJob_g * somme_g * sizeof(int));
    hipMalloc( (void**) &tempsLag_d, nbJob_g * somme_g * sizeof(int));
    hipMalloc( (void**) &nbFois_d, somme_g  * sizeof(int));
    hipMalloc( (void**) &ordoSomme_d, somme_g  * sizeof(int));
    hipMalloc( (void**) &minTempsArr_d, nbMachines_g * sizeof(int));
    hipMalloc( (void**) &minTempsDep_d, nbMachines_g * sizeof(int));
    hipMalloc( (void**) &machine_d, 2 * somme_g  * sizeof(int));
}

void gpu::copy_to_device()
{
    hipMemcpy(tempsJob_d, tempsJob_g, nbJob_g * nbMachines_g * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tabJohnson_d, tabJohnson_g, nbJob_g * somme_g * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tempsLag_d, tempsLag_g, nbJob_g * somme_g * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nbFois_d, nbFois_g, somme_g  * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ordoSomme_d, ordoSomme_g, somme_g  * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(minTempsArr_d, minTempsArr_g, nbMachines_g * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(minTempsDep_d, minTempsDep_g, nbMachines_g * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(machine_d, machine_g, 2 * somme_g  * sizeof(int), hipMemcpyHostToDevice);
}
