
using namespace std;

#include <limits.h>
#include <sys/time.h>
#include <iostream>
#include <sstream>
#include <functional>
#include <fstream>
#include <list>
#include <queue>
#include <string>
#include <pthread.h>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <sys/times.h>
#include <limits.h>
#include <algorithm>
#include <sched.h>

#include "../int/BigInteger.hh"
#include "../int/BigIntegerUtils.hh"
#include "../headers/ttime.h"
#include "../headers/decompose_beginend.h"
#include "../headers/weights.h"
#include "../headers/peer.h"
#include "../headers/problem.h"
#include "../headers/problems.h"
#include "../headers/solutions.h"
#include "../headers/work.h"
#include "../headers/instance_abstract.h"
#include "../headers/pbab.h"
#include "../headers/tree.h"
#include "../headers/explorer.h"
#include "../headers/gpu.h"
#include "../headers/works.h"

#include "../headers/work_stealing/types.h"

//============================================================================================================
//EXPLORATION===============================================================================================
//============================================================================================================

hipError_t cudaerror_t1;

void checkCUDAError1(const char *msg)
{
    hipError_t err = hipGetLastError();

    if( hipSuccess != err)
    {
    	fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

bool comparator_homogeneous::operator()(problem const* n1, problem const* n2)//return  (n1>n2)
{
    if (n1->limite2 != n2->limite2) return (n1->limite2 > n2->limite2);
    else return (n1->limite1 > n2->limite1);
}

explorer::explorer(pbab*_pbb)
{
	pbb = _pbb;

	type = arguments::type;
}

void explorer::shareBranchAndBoundProblem(){
	// if there is someone ask me for work
	if (pbb->node->pendingWorkRequest.size() > 0 && pbb->tr->babTreeForShare.size() == 0){

		long long startFoldingTime = statistics::time_get_in_us();
		// extract an interval to share to CPU
		//
		pthread_mutex_lock(&pbb->tr->bab_share_tree_mutex);

		//
		pbb->tr->gpuShareProblems();

		//
		pthread_mutex_unlock(&pbb->tr->bab_share_tree_mutex);

		long long endFoldingTime = statistics::time_get_in_us();
		statistics::total_GPU_FOLD_time += endFoldingTime - startFoldingTime;
	}
}

void explorer::_explore_cpu()
{

	int cout = 0;

	long startCPUComputingTime = statistics::time_get_in_us();

	while (true)
	{
		cout++;
		raw_bb_problem *n = pbb->tr->take();

		if (!n) break;
		
		if(this->pbb->tr->bbCalculator->leaf(n))
		{


			if (pbb->wsltns->insert(n)){
				pbb->node->sendOptimalSolutionToGPUs(pbb);
			}

			// leaves explored
			pbb->tr->cpu_explored_leaves += pbb->wghts->depths[n->depth];
		}
		else
		{

			if (pbb->wsltns->inserable(n))
			{
				pbb->tr->insert((*pbb->decompose)(*n));
				pbb->tr->cpu_explored_nodes++;
			}
			// prune
			else{
				pbb->tr->boundedNodesDontExplore++;
				pbb->tr->cpu_explored_leaves += pbb->wghts->depths[n->depth];
			}
		}

		free (n);

		/*
		 * share work request here
		 */
		if (pbb->node->pendingWorkRequest.size() > 0 || cout > 250)
			break;
	}

	long endCPUComputingTime = statistics::time_get_in_us();
		statistics::total_CPU_computing_time += endCPUComputingTime - startCPUComputingTime;


	// compute the computing capability of me
	if (pbb->tr->cpu_explored_nodes > 0)
		pbb->node->computing_capability = (double) (statistics::total_CPU_computing_time / (pbb->tr->cpu_explored_nodes));
}

void explorer::_explore_gpu(){

}

void explorer::_explore_cpu_gpu(){
	int pool_size = this->pbb->gp->pool_size;

	pbb->gp->synchronize_counter_cpu_gpu = 0;


	long startCPUComputingTime = statistics::time_get_in_us();

	/*
	 * DECOMPOSE, PREPARE POOL FOR GPU
	 */

	while (pbb->tr->size() > 0 && pbb->gp->readyPoolToGPU.size() < MAX_SIZE_INPUTQUEUE_TO_GPU){

		problems* fathers = new problems();
		bool ok;

		// Collecting problems from branch and bound tree
		while(true)
		{
			raw_bb_problem *p = pbb->tr->take();

			if (p == NULL)	{
				break;

			}else{
				if (pbb->tr->bound(p)){
					ok = fathers->push_back1(p,(pool_size - 1));
					if(!ok)
					{
						pbb->tr->insert(p);
						break;
					}
				}else{
						pbb->tr->explored_leaves += pbb->wghts->depths[p->depth];
						pbb->tr->boundedNodesDontExplore++;
						free (p);
				}
			}
		}

		//
		pbb->tr->bounded += fathers->size();
		if (fathers->size() > 0){
			// decompose the collected problems
			if (pbb->gp->decompose_on_cpu(fathers))
				pbb->gp->synchronize_counter_cpu_gpu++;
		}
		
		fathers->empty();
		delete fathers;

		if (pbb->gp->outputPoolOfGPU.size() > 0)
			break;
	}

	/*****************************************************************/
	/*
	 *  PROCESS OUTPUT OF GPU
	 */
	switch (this->pbb->cpu_operation){
		/*****/
		case CPU_WORKSTEALING:
			if (pbb->gp->readyPoolToGPU.size() > 0){
				//
				long startGPUComputingTime = statistics::time_get_in_us();

				while (!pbb->gp->readyPoolToGPU.empty()){
//					if (VERBOSE)
//						cout << "_explore_gpu: GPU readyPoolToGPU: " << pbb->gp->readyPoolToGPU.size() << endl;
					
					// Wait for the data from GPU
					cudaerror_t1 = hipStreamSynchronize(pbb->gp->stream[pbb->gp->starting_index_gpu]);
					checkCUDAError1("_explore_gpu");

					//
					problemPool* pp = pbb->gp->readyPoolToGPU.front();

					pp->children_bounds = pbb->gp->bounds_h[pbb->gp->starting_index_gpu];

					//
					pbb->gp->prune(pp);

					//
					pbb->gp->readyPoolToGPU.pop_front();

					//
					delete(pp);
					pbb->gp->starting_index_gpu = (pbb->gp->starting_index_gpu + 1) % MAX_SIZE_INPUTQUEUE_TO_GPU;


				}

				//
				long endGPUComputingTime = statistics::time_get_in_us();
				statistics::total_GPU_computing_time += endGPUComputingTime - startGPUComputingTime;
			}
			break;
		/*****/
//		case CPU_DECOMPOSE:
//
//			break;
//		case CPU_PRUNE_DECOMPOSE:
//
//			if (pbb->node->pendingWorkRequestFromMS.size() > 0 && pbb->tr->outputGPUForShare.size() == 0){
//				int count = pbb->node->pendingWorkRequestFromMS.size();
//
//				while (cout > 0 && pbb->gp->outputPoolOfGPU.size() > 0){
//					cout << "HA: " << pbb->gp->outputPoolOfGPU.size() << endl;
//					cout << "HA1: " << pbb->node->pendingWorkRequestFromMS.size() << endl;
//
//					problemPool* pp = pbb->gp->outputPoolOfGPU.front();
//					//
//					pbb->gp->prune_shareOutputGPU(pp);
//					//
//					pbb->gp->outputPoolOfGPU.pop_front();
//
//					count--;
//				}
//			}else{
//				if (pbb->gp->outputPoolOfGPU.size() > 0){
//						problemPool* pp = pbb->gp->outputPoolOfGPU.front();
//						pbb->gp->outputPoolOfGPU.pop_front();
//
//						//
//						pbb->gp->prune(pp);
//
//						//
//						free(pp->children_d);
//						free(pp->children_bounds);
//						delete pp;
//				}
//			}
//			break;
//		case CPU_NORMAL_EXPLORE: break;
//		case CPU_ADAPTIVE: break;
//		default: cout << "CPU operation NOT_DEFINED" << endl; break;
	}

	//
	long endCPUComputingTime = statistics::time_get_in_us();
	statistics::total_CPU_computing_time += endCPUComputingTime - startCPUComputingTime;

	// compute the computing capability of me
	if (pbb->tr->bounded > 0)
		pbb->node->computing_capability = (double) (statistics::total_CPU_computing_time / (pbb->tr->bounded));

	/*****************************************************************/
	/*
	 * SHARE WORKS TO OTHERS
	 */
}

void * tree_thread(void *_e)
{
	explorer *e = (explorer*) _e;

	if (e->type == 'c')	e->_explore_cpu();
	else if (e->type == 'g')	e->_explore_gpu();

		else std::cout<<"ERROR in tree thread, no CPU, and GPU "<<flush;

	return NULL;
}


void explorer::explore()
{
	tree_thread(this);
}

void explorer::run()
{
	if (arguments::worker)	pbb->xplr->explore();
	else
	{
		pbb->wrks->cout = arguments::costv;
		pbb->init();
		sleep(99999999); //attendre lorsqu'il s'agit du coordinateur
	}
}

void explorer::cpu_decompose_subpblems(problems* fathers, raw_bb_problem **p, int* numOfProblems){
	cout << statistics::time_get_in_second() << ": decompose_on_cpu, parent size: " << fathers->size() << endl;
	cout << statistics::time_get_in_second() << ": decompose_on_cpu, parent size1: " << fathers->size1 << endl;

	raw_bb_problem* tmp = (raw_bb_problem *) malloc(fathers->size1 * sizeof(raw_bb_problem)) ;
	raw_bb_problem* tmp1 = tmp;

	int index = -1;

	for (int l = 0; l < fathers->size() ; l++)
	{
		if (this->pbb->tr->bbCalculator->simple(fathers->at(l)))
			pbb->decompose->leaves_generate3(fathers->at(l),tmp1, &index);
		else
		{
			pbb->decompose->problems_generate3(tmp1,fathers->at(l),BEGIN_ORDER, &index);

			pbb->decompose->problems_generate3(tmp1,fathers->at(l),END_ORDER, &index);
		}

	}

	if (index >= 0){
		*p = tmp;
		*numOfProblems = index + 1;
	}else {
		free(tmp);
		*p = NULL;
		*numOfProblems = 0;
	}
}

/*
 * CPU decompose bab problems and send the results to GPU
 */
void explorer::cpu_explore_decomposing(){

	int pool_size = this->pbb->gp->pool_size;;

	problems* fathers = new problems();
	bool ok;

	while (true){
		raw_bb_problem *n = pbb->tr->take();

		if (!n) break;

		if(this->pbb->tr->bbCalculator->leaf(n))
		{
			if (pbb->tr->bound(n))
				pbb->wsltns->insert(n);
				pbb->node->sendOptimalSolutionToGPUs(pbb);

			// leaves explored
			pbb->tr->cpu_explored_leaves += pbb->wghts->depths[n->depth];
			free(n);
		}
		else
		{
			/*
			 *  prepare to decompose here
			 */
			if (pbb->tr->bound(n))
			{
				ok = fathers->push_back1(n,(pool_size - 1));
				if(!ok)
				{
					pbb->tr->insert(n);
					break;
				}

				pbb->tr->cpu_explored_nodes += fathers->size();
			}
			// prune
			else{
				pbb->tr->cpu_explored_leaves += pbb->wghts->depths[n->depth];
				free(n);
			}
		}
	}
	/*
	 *  decompose the collected problems
	 */
	if (fathers->size() > 0){
		raw_bb_problem* pp = NULL;
		int sizeInElements;
		sizeInElements = 0;

		this->cpu_decompose_subpblems(fathers, &pp, &sizeInElements);
		cout << "AAAAA, sizeInElements: " << sizeInElements << endl;

		// send the decomposed sub problems to GPU
		if (sizeInElements > 0 & pp != NULL){
			pbb->node->sendDecomposedSubProblemsToGPU(sizeInElements, (char*) pp);
			free(pp);
		}else {
			free(pp);
		}

	}
	fathers->empty();
	delete fathers;

}

void explorer::cpu_pruneDecomposeProblem(){
	while (this->pbb->tr->babProblemsForPrune.size() > 0 || this->pbb->tr->size() > 0) {
		// Prune
		while (this->pbb->tr->babProblemsForPrune.size() > 0){
			cpu_raw_bb_problem* recv_problems = this->pbb->tr->babProblemsForPrune.front();

			int num_problems = recv_problems->size_int_byte / sizeof (raw_bb_problem);
			this->pbb->gp->prune(recv_problems->data, num_problems);

			this->pbb->tr->babProblemsForPrune.pop_front();
			free(recv_problems->data);
			free(recv_problems);
		}

		// Decompose + Return the decompose one to GPU
		cpu_explore_decomposing();


	}
}
