#include "hip/hip_runtime.h"
void initSomme_g() {
	somme_g = 0;

	for (int i = 1; i < nbMachines_g; i++)
		somme_g += i;
}

void free_memories() {
	for (int i = 0; i < nbMachines_g; i++)
		free(tempsFinMachines_g[i]);

	free(tempsFinMachines_g);

	for (int i = 0; i < somme_g; i++)
		free(tempsEntre_g[i]);

	free(tempsEntre_g);
	free(tempsFin_g);
	free(tempsExeTrie_g);
	free(tempsEntreTrie_g);
	free(tempsFinMachinesTrie_g);
	free(tempsLag_g);
	free(ordoSomme_g);
	free(nbFois_g);
	free(tempsFinTrie_g);
	free(tabJohnson_g);
	free(machine_g);
	free(tempsArriver_g);
	free(tempsDepart_g);
	free(minTempsArr_g);
	free(minTempsDep_g);
	free(tempsJob_g);
}

void allouerMemoire_g() {
	tempsFin_g = (int *) malloc(nbJob_g * sizeof(int));

	tempsExeTrie_g = (int *) malloc(nbMachines_g * nbJob_g * sizeof(int));

	tempsFinMachines_g = (int **) malloc(nbMachines_g * sizeof(int *));

	for (int i = 0; i < nbMachines_g; i++)
		tempsFinMachines_g[i] = (int *) malloc(nbJob_g * sizeof(int));

	tempsEntre_g = (int **) malloc(somme_g * sizeof(int *));

	for (int i = 0; i < somme_g; i++)
		tempsEntre_g[i] = (int *) malloc(nbJob_g * sizeof(int));

	tempsEntreTrie_g = (int *) malloc(somme_g * nbJob_g * sizeof(int));
	tempsFinMachinesTrie_g = (int *) malloc(
			nbMachines_g * nbJob_g * sizeof(int));
	tempsLag_g = (int *) malloc(somme_g * nbJob_g * sizeof(int));

	ordoSomme_g = (int *) malloc(somme_g * sizeof(int));
	nbFois_g = (int *) malloc(somme_g * sizeof(int));

	tempsFinTrie_g = (int *) malloc(nbJob_g * sizeof(int));

	tabJohnson_g = (int *) malloc(somme_g * nbJob_g * sizeof(int));

	machine_g = (int *) malloc(2 * somme_g * sizeof(int));

	tempsArriver_g = (int *) malloc(nbMachines_g * nbJob_g * sizeof(int));
	tempsDepart_g = (int *) malloc(nbMachines_g * nbJob_g * sizeof(int));

	minTempsArr_g = (int *) malloc(nbMachines_g * sizeof(int));
	minTempsDep_g = (int *) malloc(nbMachines_g * sizeof(int));
}

long double aFaire_g(int profondeur) {
	long double nbNoeudAFaire = 1;

	for (int i = 2; i <= profondeur; i++)
		nbNoeudAFaire = (nbNoeudAFaire * i) + i;

	return nbNoeudAFaire;
}

void initTempsExeTrie_g() {
	for (int i = 0; i < nbMachines_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tempsExeTrie_g[i * nbJob_g + j] = j;
}

void initTempsEntreTrie_g() {
	for (int i = 0; i < somme_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tempsEntreTrie_g[i * nbJob_g + j] = j;
}

void initTempArrays_g() {
	tempsExeTrie_T = (int **) malloc(nbMachines_g * sizeof(int *));
	tempsFinMachinesTrie_T = (int **) malloc(nbMachines_g * sizeof(int *));

	for (int i = 0; i < nbMachines_g; i++) {
		tempsExeTrie_T[i] = (int *) malloc(nbJob_g * sizeof(int));
		tempsFinMachinesTrie_T[i] = (int *) malloc(nbJob_g * sizeof(int));
	}

	tempsEntreTrie_T = (int **) malloc(somme_g * sizeof(int *));
	tabJohnson_T = (int **) malloc(somme_g * sizeof(int *));

	for (int i = 0; i < somme_g; i++) {
		tempsEntreTrie_T[i] = (int *) malloc(nbJob_g * sizeof(int));
		tabJohnson_T[i] = (int *) malloc(nbJob_g * sizeof(int));
	}

	for (int i = 0; i < somme_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tempsEntreTrie_T[i][j] = j;

	for (int i = 0; i < nbMachines_g; i++)
		for (int j = 0; j < nbJob_g; j++) {
			tempsExeTrie_T[i][j] = j;
			tempsFinMachinesTrie_T[i][j] = j;
		}
}

void initTempsFinTrie_g() {
	for (int i = 0; i < nbJob_g; i++)
		tempsFinTrie_g[i] = i;
}

void initTempsFinMachinesTrie_g() {
	for (int i = 0; i < nbMachines_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tempsFinMachinesTrie_g[i * nbJob_g + j] = j;
}

int estInf_g(int i, int j) {
	if (pluspetit_g[0][i] == pluspetit_g[0][j]) {
		if (pluspetit_g[0][i] == 1)
			return pluspetit_g[1][i] < pluspetit_g[1][j];

		return pluspetit_g[1][i] > pluspetit_g[1][j];
	}

	return pluspetit_g[0][i] < pluspetit_g[0][j];
}

int estSup_g(int i, int j) {
	if (pluspetit_g[0][i] == pluspetit_g[0][j]) {
		if (pluspetit_g[0][i] == 1)
			return pluspetit_g[1][i] > pluspetit_g[1][j];

		return pluspetit_g[1][i] < pluspetit_g[1][j];
	}

	return pluspetit_g[0][i] > pluspetit_g[0][j];
}

int partionner_g(int *ordo, int deb, int fin) {
	int d = deb - 1;
	int f = fin + 1;
	int mem, pivot = ordo[deb];

	do {

		do {
			f--;
		} while (estSup_g(ordo[f], pivot));

		do {
			d++;
		} while (estInf_g(ordo[d], pivot));

		if (d < f) {
			mem = ordo[d];
			ordo[d] = ordo[f];
			ordo[f] = mem;
		}

	} while (d < f);

	return f;
}

void quicksort_g(int *ordo, int deb, int fin) {
	int k;

	if ((fin - deb) > 0) {
		k = partionner_g(ordo, deb, fin);
		quicksort_g(ordo, deb, k);
		quicksort_g(ordo, k + 1, fin);
	}
}

void trierTableau_g(int *ordo, int nbElem_g, int *nbFois_g, bool croissant) {
	pluspetit_g[0] = (int *) malloc((nbElem_g) * sizeof(int));
	pluspetit_g[1] = (int *) malloc((nbElem_g) * sizeof(int));

	for (int i = 0; i < nbElem_g; i++) {
		if (croissant)
			pluspetit_g[0][i] = 1;
		else
			pluspetit_g[0][i] = 2;

		pluspetit_g[1][i] = nbFois_g[i];
	}

	quicksort_g(ordo, 0, (nbElem_g - 1));

	free(pluspetit_g[0]);
	free(pluspetit_g[1]);
}

void remplirTempsExeTrie_g() {
	for (int i = 0; i < nbMachines_g; i++)
		trierTableau_g(tempsExeTrie_T[i], nbJob_g, tempsJob_T[i], true);

	for (int i = 0; i < nbMachines_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tempsExeTrie_g[i * nbJob_g + j] = tempsExeTrie_T[i][j];

	for (int i = 0; i < nbMachines_g; i++)
		free(tempsExeTrie_T[i]);

	free(tempsExeTrie_T);
}

void remplirTempsArriverDepart_g() {
	bool faitDep[nbMachines_g];
	bool faitArr[nbMachines_g];

	int machine_g;

	for (int k = 1; k < nbMachines_g; k++) {
		faitArr[k] = false;
		faitDep[k] = false;
	}

	minTempsDep_g[nbMachines_g - 1] = 0;
	minTempsArr_g[0] = 0;

	for (int i = 0; i < nbJob_g; i++) {
		tempsArriver_g[i] = 0;
		tempsDepart_g[(nbMachines_g - 1) * nbJob_g + i] = 0;

		for (int k = 1; k < nbMachines_g; k++) {
			machine_g = nbMachines_g - k - 1;
			tempsArriver_g[k * nbJob_g + i] = tempsArriver_g[(k - 1) * nbJob_g
					+ i] + tempsJob_g[(k - 1) * nbJob_g + i];
			tempsDepart_g[machine_g * nbJob_g + i] = tempsDepart_g[(machine_g
					+ 1) * nbJob_g + i]
					+ tempsJob_g[(machine_g + 1) * nbJob_g + i];

			if (!faitArr[k]
					|| minTempsArr_g[k] > tempsArriver_g[k * nbJob_g + i]) {
				faitArr[k] = true;
				minTempsArr_g[k] = tempsArriver_g[k * nbJob_g + i];
			}

			if (!faitDep[k]
					|| minTempsDep_g[machine_g]
							> tempsDepart_g[machine_g * nbJob_g + i]) {
				faitDep[k] = true;
				minTempsDep_g[machine_g] =
						tempsDepart_g[machine_g * nbJob_g + i];
			}
		}
	}
}

void remplirMachine_g() {
	int cmpt = 0;

	for (int i = 0; i < (nbMachines_g - 1); i++)
		for (int j = i + 1; j < nbMachines_g; j++) {
			machine_g[cmpt] = i;
			cmpt++;
		}

	int taille = cmpt;
	cmpt = 0;

	for (int i = 0; i < (nbMachines_g - 1); i++)
		for (int j = i + 1; j < nbMachines_g; j++) {
			machine_g[1 * taille + cmpt] = j;
			cmpt++;
		}
}

void remplirLag_g() {
	int m1, m2;

	for (int i = 0; i < somme_g; i++) {
		m1 = machine_g[0 * somme_g + i];
		m2 = machine_g[1 * somme_g + i];

		for (int j = 0; j < nbJob_g; j++) {
			tempsLag_g[i * nbJob_g + j] = 0;

			for (int k = m1 + 1; k < m2; k++)
				tempsLag_g[i * nbJob_g + j] += tempsJob_g[k * nbJob_g + j];

			tempsEntre_g[i][j] = tempsLag_g[i * nbJob_g + j]
					+ tempsJob_g[m2 * nbJob_g + j];
		}
	}
}

void remplirTempsEntreTrie_g() {
	for (int i = 0; i < somme_g; i++)
		trierTableau_g(tempsEntreTrie_T[i], nbJob_g, tempsEntre_g[i], true);

	for (int i = 0; i < somme_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tempsEntreTrie_g[i * nbJob_g + j] = tempsEntreTrie_T[i][j];

	for (int i = 0; i < somme_g; i++)
		free(tempsEntreTrie_T[i]);

	free(tempsEntreTrie_T);
}

void Johnson_g(int *ordo, int m1, int m2, int s) {
	pluspetit_g[0] = (int *) malloc(nbJob_g * sizeof(int));
	pluspetit_g[1] = (int *) malloc(nbJob_g * sizeof(int));

	for (int i = 0; i < nbJob_g; i++) {
		ordo[i] = i;

		if (tempsJob_g[m1 * nbJob_g + i] < tempsJob_g[m2 * nbJob_g + i]) {
			pluspetit_g[0][i] = 1;
			pluspetit_g[1][i] = tempsJob_g[m1 * nbJob_g + i]
					+ tempsLag_g[s * nbJob_g + i];
		} else {
			pluspetit_g[0][i] = 2;
			pluspetit_g[1][i] = tempsJob_g[m2 * nbJob_g + i]
					+ tempsLag_g[s * nbJob_g + i];
		}
	}

	quicksort_g(ordo, 0, (nbJob_g - 1));

	free(pluspetit_g[0]);
	free(pluspetit_g[1]);
}

void remplirTabJohnson_g() {
	int cmpt = 0;

	for (int i = 0; i < (nbMachines_g - 1); i++)
		for (int j = i + 1; j < nbMachines_g; j++) {
			Johnson_g(tabJohnson_T[cmpt], i, j, cmpt);
			cmpt++;
		}

	for (int i = 0; i < somme_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tabJohnson_g[i * nbJob_g + j] = tabJohnson_T[i][j];

	for (int i = 0; i < somme_g; i++)
		free(tabJohnson_T[i]);

	free(tabJohnson_T);
}

void remplirTempsFinMachines_g() {
	for (int i = 0; i < nbJob_g; i++)
		tempsFinMachines_g[nbMachines_g - 1][i] = tempsFin_g[i];

	for (int i = nbMachines_g - 2; i >= 0; i--)
		for (int j = 0; j < nbJob_g; j++)
			tempsFinMachines_g[i][j] = tempsFinMachines_g[i + 1][j]
					- tempsJob_T[i + 1][j];

	for (int i = 0; i < nbMachines_g; i++)
		free(tempsJob_T[i]);

	free(tempsJob_T);
}

void remplirTempsFinMachinesTrie_g() {
	for (int i = 0; i < nbMachines_g; i++)
		trierTableau_g(tempsFinMachinesTrie_T[i], nbJob_g,
				tempsFinMachines_g[i], true);

	for (int i = 0; i < nbMachines_g; i++)
		for (int j = 0; j < nbJob_g; j++)
			tempsFinMachinesTrie_g[i * nbJob_g + j] =
					tempsFinMachinesTrie_T[i][j];

	for (int i = 0; i < nbMachines_g; i++)
		free(tempsFinMachinesTrie_T[i]);

	free(tempsFinMachinesTrie_T);
}

void initNbFois_g() {
	for (int i = 0; i < somme_g; i++) {
		ordoSomme_g[i] = i;
		nbFois_g[i] = 0;
	}
}

void initialiserVar_g() {
	initSomme_g();
	allouerMemoire_g();
	initTempArrays_g();
	initTempsFinTrie_g();
	initTempsExeTrie_g();
	initTempsEntreTrie_g();
	initTempsFinMachinesTrie_g();
	trierTableau_g(tempsFinTrie_g, nbJob_g, tempsFin_g, true);
	remplirTempsExeTrie_g();
	remplirTempsArriverDepart_g();
	remplirMachine_g();
	remplirLag_g();
	remplirTempsEntreTrie_g();
	remplirTabJohnson_g();
	remplirTempsFinMachines_g();
	remplirTempsFinMachinesTrie_g();
	initNbFois_g();
}

void init_bound() {
	nbOrdo_g = 1;
	nbborne_g = 1;
	nbNoeud_g = 0;
	nbPartition_g = 10;
	nbRetardNonAffB1_g = 0;
	nbRetardNonAffB2_g = 0;
	nbZero_g = 0;
	seuil_g = 0;

	initialiserVar_g();

	nbElem_g = somme_g;

	nbNoeudTotal_g = aFaire_g(nbJob_g);
}

void init_problem_device_g(problem_d &p) {
	p.limite1 = -1;
	p.limite2 = nbJob_g;

	for (int j = 0; j < nbJob_g; j++)
		p.permutation[j] = j;

	p.couts_somme = 0;

	p.depth = 0;
}

inline __device__ void set_job_jobFin_g(int* job, int* jobFin, int* permutation, int limite1, int limite2,int nbJob_g)
{
	int j = 0;

	for(j = 0;j <= limite1;j++) job[permutation[j]] = j + 1;

	for(j = limite1 + 1;j < limite2;j++) job[permutation[j]] = 0;

	for(j = limite2; j < nbJob_g;j++)
	{
		job[permutation[j]] = j + 1;
		jobFin[j] = permutation[j];
	}
}

inline __device__ int cmaxFin_g(int *tempsMachinesFin, int *tmp, int *ma)
{
	return max(tmp[1] + tempsMachinesFin[ma[1]],tmp[0] + tempsMachinesFin[ma[0]]);
}

inline __device__ void initCmax_g(int* tempsMachines,int nbAffectDebut, int *tmp, int *ma, int ind,int * machine_g, int somme_g, int * minTempsArr_g)
{
	ma[0] = machine_g[ind];
	ma[1] = machine_g[1 * somme_g + ind];

	int coeff = __cosf(nbAffectDebut);
	tmp[0] = (1 - coeff) * tempsMachines[ma[0]] + coeff * minTempsArr_g[ma[0]];
	tmp[1] = (1 - coeff) * tempsMachines[ma[1]] + coeff * minTempsArr_g[ma[1]];
}

inline __device__ void heuristiqueCmax_g(int* job, int *tmp, int *ma, int ind, int nbJob_g, int * tabJohnson_g, unsigned char* tempsJob_g,int * tempsLag_g)
{
	int jobCour;

	for(int j= 0; j < nbJob_g; j++)
	{
		jobCour = tabJohnson_g[ind * nbJob_g + j];

		if( job[jobCour] == 0 )
		{
			tmp[0] = tmp[0] + tempsJob_g[ma[0] * nbJob_g + jobCour];
			tmp[1] = max (tmp[1], tmp[0] + tempsLag_g[ind * nbJob_g + jobCour]) + tempsJob_g[ma[1] * nbJob_g + jobCour];
		}
	}
}

inline __device__ int criteres_calculer_g(int* permutation,int nbMachines_g,int nbJob_g,unsigned char* tempsJob_g)
{
	int temps[MAX_NB_MACHINES];

	for(int mm = 0; mm < nbMachines_g; mm++) temps[mm] = 0;

	for(int j = 0; j < nbJob_g;j++)
	{
		int job = permutation[j];

		temps[0] = temps[0] + tempsJob_g[job];

		for(int m = 1; m < nbMachines_g;m++)
		temps[m] = max(temps[m],temps[m-1]) + tempsJob_g[m * nbJob_g + job];
	}

	return temps[nbMachines_g-1];
}

inline __device__ void set_tempsMachines_retardDebut_g(int *tempsMachines, int* permutation, int limite1,int nbMachines_g,int nbJob_g,unsigned char* tempsJob_g)
{
	int m = 0;

	for(m = 0; m < nbMachines_g; m++) tempsMachines[m] = 0;

	for(int j = 0; j <= limite1; j++)
	{
		int job = permutation[j];

		tempsMachines[0] = tempsMachines[0] + tempsJob_g[job];

		for(m = 1; m < nbMachines_g;m++)
		tempsMachines[m] = max(tempsMachines[m],tempsMachines[m-1]) + tempsJob_g[m * nbJob_g + job];
	}
}

inline __device__ void set_tempsMachinesFin_tempsJobFin_g(int* jobFin, int * tempsMachinesFin,int nbAffectFin,int nbJob_g, int nbMachines_g,unsigned char* tempsJob_g)
{
	int jobCour;

	int tmpMa[MAX_NB_MACHINES];

	for(int j = 0; j < nbMachines_g; j++)
	{
		for(int k = j; k < nbMachines_g; k++) tmpMa[k] = 0;

		for(int k = nbJob_g - nbAffectFin; k < nbJob_g; k++)
		{
			jobCour = jobFin[k];

			tmpMa[j] = tmpMa[j] + tempsJob_g[j * nbJob_g + jobCour];

			for(int l = j + 1; l < nbMachines_g; l++)
			{
				tmpMa[l] = max (tmpMa[l-1],tmpMa[l]);
				tmpMa[l] = tmpMa[l] + tempsJob_g[l * nbJob_g + jobCour];
			}
		}

		tempsMachinesFin[j] = tmpMa[nbMachines_g-1];
	}
}

inline __device__ int borneInfMakespan_g(int* job,int *tempsMachinesFin,int* tempsMachines,
		int nbAffectDebut,int nbAffectFin,int *valBorneInf, int retardNonFin, int minCmax,int nbJob_g, int nbElem_g,
		int leMeilleur_g, int nbborne_g, int somme_g, int * minTempsArr_g, int*nbFois_g, int *machine_g, int * tabJohnson_g,
		unsigned char* tempsJob_g, int* tempsLag_g, int* ordoSomme_g, int* minTempsDep_g)
{
	int moinsBon = 0;

	int idxMoinsBonCmax, i, tmpDep, retard = retardNonFin;

	int ma[2] = {0,0};
	int tmp[2] = {0,0};
	int res[2] = {0,0};

	int l;

	int coeffReturn=1;

	for(l = 0; l < nbElem_g; l++)
	{
		i = ordoSomme_g[l];
		initCmax_g(tempsMachines,nbAffectDebut,tmp,ma,i,machine_g,somme_g,minTempsArr_g);
		heuristiqueCmax_g(job, tmp, ma, i, nbJob_g,tabJohnson_g, tempsJob_g, tempsLag_g);

		if( nbAffectFin != 0 ) tmp[1] = cmaxFin_g(tempsMachinesFin, tmp, ma);
		else tmp[1] = tmp[1] + minTempsDep_g[ma[1]];

		float un = 1;
		int coeff2 = min(un, __expf(tmp[1] - moinsBon - 1));
		idxMoinsBonCmax = coeff2 * i;
		moinsBon = max(moinsBon,tmp[1]);
		int coeff3 = min(un,__expf(moinsBon - leMeilleur_g - 1));
		int coeff4 = min(1,(leMeilleur_g + 2) ^ 1);
		int coeff5 = coeff3 * coeff4;

		nbborne_g = nbborne_g + coeff5;

		valBorneInf[0] = valBorneInf[0] + coeff5 * moinsBon;
		coeffReturn = coeff5;
	}

	nbborne_g++;
	nbFois_g[idxMoinsBonCmax]++;
	valBorneInf[0] = moinsBon;

	return 0;
}

inline __device__ int calculBorne_g(int* job,int *tempsMachinesFin,int *tempsMachines,int nbAffectDebut,int nbAffectFin,int nbJob_g,int leMeilleur_g, int nbborne_g,int somme_g,int nbElem_g,unsigned char* tempsJob_g,int* nBfois,int* machine_g,int* tabJohnson_g,int* tempsLag_g,int * minTempsArr_g,int* ordoSomme_g,int* minTempsDep_g)
{
	int retardNonAff = 0;

	int minCmax = 0;

	int valBorneInf[2];

	int retardNonFin = retardNonAff;

	borneInfMakespan_g(job,tempsMachinesFin,tempsMachines,nbAffectDebut,nbAffectFin,valBorneInf,retardNonFin,minCmax,nbJob_g,nbElem_g,leMeilleur_g,nbborne_g,somme_g,minTempsArr_g,
			nBfois,machine_g,tabJohnson_g,tempsJob_g,tempsLag_g,ordoSomme_g,minTempsDep_g);

	return valBorneInf[0];
}

__global__ void Evaluate_ON_GPU(raw_bb_problem* pool_to_evaluate, int* bounds,
		int nbJob_g, int nbMachines_g, int nbborne_g, int somme_g, int nbElem_g,
		int *nbFois_g, int *machine_g, int *tabJohnson_g, int *tempsJob_g,
		int *tempsLag_g, int *minTempsArr_g, int *ordoSomme_g,
		int *minTempsDep_g, int todo, int time_seed_g) {
	int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__
	unsigned char tempsJob_g_shared[MAX_NB_MACHINES * MAX_NBJOBS];

	if (thread_idx < todo) {
		if (threadIdx.x == 0) {
			for (int i = 0; i < nbMachines_g; i++)
				for (int j = 0; j < nbJob_g; j++)
					tempsJob_g_shared[i * nbJob_g + j] = tempsJob_g[i * nbJob_g
							+ j];
		}

		__syncthreads();

		int tempsMachines[MAX_NB_MACHINES];
		int tempsMachinesFin[MAX_NB_MACHINES];

		int job[MAX_NBJOBS];
		int jobFin[MAX_NBJOBS];
		int nbAffectFin = nbJob_g - pool_to_evaluate[thread_idx].limite2;
		int nbAffectDebut = pool_to_evaluate[thread_idx].limite1 + 1;
		int leMeilleur_g = 999999;
		int borneInf = 0;

		if (pool_to_evaluate[thread_idx].limite2
				- pool_to_evaluate[thread_idx].limite1 == 1)
			borneInf = criteres_calculer_g(
					pool_to_evaluate[thread_idx].permutation, nbMachines_g,
					nbJob_g, tempsJob_g_shared);
		else {
			set_tempsMachines_retardDebut_g(tempsMachines,
					pool_to_evaluate[thread_idx].permutation,
					pool_to_evaluate[thread_idx].limite1, nbMachines_g, nbJob_g,
					tempsJob_g_shared);
			set_job_jobFin_g(job, jobFin,
					pool_to_evaluate[thread_idx].permutation,
					pool_to_evaluate[thread_idx].limite1,
					pool_to_evaluate[thread_idx].limite2, nbJob_g);
			set_tempsMachinesFin_tempsJobFin_g(jobFin, tempsMachinesFin,
					nbAffectFin, nbJob_g, nbMachines_g, tempsJob_g_shared);
			borneInf = calculBorne_g(job, tempsMachinesFin, tempsMachines,
					nbAffectDebut, nbAffectFin, nbJob_g, leMeilleur_g,
					nbborne_g, somme_g, nbElem_g, tempsJob_g_shared, nbFois_g,
					machine_g, tabJohnson_g, tempsLag_g, minTempsArr_g,
					ordoSomme_g, minTempsDep_g);
		}

		bounds[thread_idx] = borneInf;
	}
}
